#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <sys/timeb.h>

#include "config.h"
#include "util.h"

__device__ void set_pixel(unsigned char* image, int width, int x, int y, unsigned char* c)
{
	image[4 * width * y + 4 * x + 0] = c[0];
	image[4 * width * y + 4 * x + 1] = c[1];
	image[4 * width * y + 4 * x + 2] = c[2];
	image[4 * width * y + 4 * x + 3] = 255;
}

__global__ void kernel(unsigned char* image, unsigned char* colormap) 
{
	int row, col, index, iteration;
	double c_re, c_im, x, y, x_new;

	int width = WIDTH;
	int height = HEIGHT;
	int maxIterations = MAX_ITERATION;
	index = blockIdx.x * blockDim.x + threadIdx.x;
	row = index / width;
	col = index % width;

	if (row >= height || col >= width || index >= LENGTH) 
	{
		return;
	}
	
	c_re = (col - width / 2.0) * 4.0 / width;
	c_im = (row - height / 2.0) * 4.0 / width;
	x = 0, y = 0;
	iteration = 0;
	while (x * x + y * y <= 4 && iteration < maxIterations) 
	{
		x_new = x * x - y * y + c_re;
		y = 2 * x * y + c_im;
		x = x_new;
		iteration++;
	}
	if (iteration > maxIterations) {
		iteration = maxIterations;
	}
	set_pixel(image, width, col, row, &colormap[iteration * 3]);
}

void onFail(const hipError_t& err, const char message[])
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "%s, Error: %s\n" ,message ,hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
int main(void)
{
	unsigned char* colormapCuda, *imageCuda;
	double times[REPEAT];
	struct timeb start, end;
	int i, r;
	char path[255];

	int colormapSize = (MAX_ITERATION + 1) * 3;
	int imageSize = LENGTH * 4;

	unsigned char* colormap = (unsigned char*)malloc(colormapSize);
	unsigned char* image = (unsigned char*)malloc(imageSize);

	init_colormap(MAX_ITERATION, colormap);

	hipError_t err = hipSuccess;

	err = hipSetDevice(0);
	onFail(err, "Failed to set device!");

	//int ndev;
	//hipDeviceProp_t p;
	//err = hipGetDeviceCount(&ndev);
	//onFail(err, "Failed to hipGetDeviceCount!");
	//for (i = 0; i < ndev; i++)
	//{
	//	err = hipGetDeviceProperties(&p, i);
	//	onFail(err, "Failed hipGetDeviceProperties!");
	//	printf("Name: %s\n", p.name);
	//	printf("Compute capability: %d.%d\n", p.major, p.minor);
	//	printf("Max threads/block: %d\n", p.maxThreadsPerBlock);
	//	printf("Max block size: %d x %d x %d\n", p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
	//	printf("Max grid size:  %d x %d x %d\n", p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
	//	printf("Warp size: %d\n", p.warpSize);
	//}

	for (r = 0; r < REPEAT; r++)
	{
		memset(image, 0, imageSize);
		ftime(&start);

		err = hipMalloc(&colormapCuda, colormapSize * sizeof(unsigned char));
		onFail(err, "Failed to hipMalloc colormapCuda!");

		err = hipMalloc(&imageCuda, imageSize * sizeof(unsigned char));
		onFail(err, "Failed to hipMalloc imageCuda!");

		err = hipMemcpy(colormapCuda, colormap, colormapSize * sizeof(unsigned char), hipMemcpyHostToDevice);
		onFail(err, "Failed to hipMemcpy hostToDevice colormapCuda!");

		err = hipMemcpy(imageCuda, image, imageSize * sizeof(unsigned char), hipMemcpyHostToDevice);
		onFail(err, "Failed to hipMemcpy hostToDevice imageCuda!");

		std::cout << "kernel " << GRID_SIZE << ", " << BLOCK_SIZE << " LEN: " << LENGTH <<"\n";
		kernel << <GRID_SIZE, BLOCK_SIZE >> > (imageCuda, colormapCuda);
		
		err = hipGetLastError();
		onFail(err, "Failed to launch kernel!");

		err = hipDeviceSynchronize();
		onFail(err, "Failed to synchronize");

		err = hipMemcpy(image, imageCuda, imageSize * sizeof(unsigned char), hipMemcpyDeviceToHost);
		onFail(err, "Failed to hipMemcpy deviceToHost imageCuda!");

		hipFree(colormapCuda);
		hipFree(imageCuda);

		ftime(&end);
		times[r] = end.time - start.time + ((double)end.millitm - (double)start.millitm) / 1000.0;

		sprintf(path, IMAGE, "gpu", r);
		save_image(path, image, WIDTH, HEIGHT);
		progress("gpu", r, times[r]);
	}
	report("gpu", times);

	err = hipDeviceReset();
	onFail(err, "Failed to hipDeviceReset!");

	free(image);
	free(colormap);
	hipFree(colormapCuda);
	hipFree(imageCuda);
	return 0;
}